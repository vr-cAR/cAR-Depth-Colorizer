#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <iostream>
#include <cstdio>
#include <algorithm>
#include <math.h>

// const float DMIN = 250;
// const float DMAX = 8000;
// const float DMIN = 5.0 / 65535.0;
// const float DMAX = 10000.0 / 65535.0;
// const float DMIN = 0.3;
// const float DMAX = 16.0;

//JOFWJEFOIWEJF
typedef struct {
    double r;       // a fraction between 0 and 1
    double g;       // a fraction between 0 and 1
    double b;       // a fraction between 0 and 1
} rgb;

typedef struct {
    double h;       // angle in degrees
    double s;       // a fraction between 0 and 1
    double v;       // a fraction between 0 and 1
} hsv;

static hsv   rgb2hsv(rgb in);
static rgb   hsv2rgb(hsv in);




hsv rgb2hsv(rgb in)
{
    hsv         out;
    double      min, max, delta;

    min = in.r < in.g ? in.r : in.g;
    min = min  < in.b ? min  : in.b;

    max = in.r > in.g ? in.r : in.g;
    max = max  > in.b ? max  : in.b;

    out.v = max;                                // v
    delta = max - min;
    if (delta < 0.00001)
    {
        out.s = 0;
        out.h = 0; // undefined, maybe nan?
        return out;
    }
    if( max > 0.0 ) { // NOTE: if Max is == 0, this divide would cause a crash
        out.s = (delta / max);                  // s
    } else {
        // if max is 0, then r = g = b = 0              
        // s = 0, h is undefined
        out.s = 0.0;
        out.h = NAN;                            // its now undefined
        return out;
    }
    if( in.r >= max )                           // > is bogus, just keeps compilor happy
        out.h = ( in.g - in.b ) / delta;        // between yellow & magenta
    else
    if( in.g >= max )
        out.h = 2.0 + ( in.b - in.r ) / delta;  // between cyan & yellow
    else
        out.h = 4.0 + ( in.r - in.g ) / delta;  // between magenta & cyan

    out.h *= 60.0;                              // degrees

    if( out.h < 0.0 )
        out.h += 360.0;

    return out;
}
rgb hsv2rgb(hsv in)
{
    double      hh, p, q, t, ff;
    long        i;
    rgb         out;

    if(in.s <= 0.0) {       // < is bogus, just shuts up warnings
        out.r = in.v;
        out.g = in.v;
        out.b = in.v;
        return out;
    }
    hh = in.h;
    if(hh >= 360.0) hh = 0.0;
    hh /= 60.0;
    i = (long)hh;
    ff = hh - i;
    p = in.v * (1.0 - in.s);
    q = in.v * (1.0 - (in.s * ff));
    t = in.v * (1.0 - (in.s * (1.0 - ff)));

    switch(i) {
    case 0:
        out.r = in.v;
        out.g = t;
        out.b = p;
        break;
    case 1:
        out.r = q;
        out.g = in.v;
        out.b = p;
        break;
    case 2:
        out.r = p;
        out.g = in.v;
        out.b = t;
        break;

    case 3:
        out.r = p;
        out.g = q;
        out.b = in.v;
        break;
    case 4:
        out.r = t;
        out.g = p;
        out.b = in.v;
        break;
    case 5:
    default:
        out.r = in.v;
        out.g = p;
        out.b = q;
        break;
    }
    return out;     
}

struct colorize_functor
{
    const float DMIN;
    const float DMAX;
    colorize_functor(float DMIN, float DMAX) : DMIN(DMIN), DMAX(DMAX) {}

    __host__ __device__
        int operator()(const short& depth) const { 
            float d = depth;
            if (depth < DMIN) d = DMIN;
            if (depth > DMAX) d = DMAX;
            float disp = 1.0 / d;
            float disp_max = 1.0 / DMIN;
            float disp_min = 1.0 / DMAX;
            float dnormal = (disp - disp_min) / (disp_max - disp_min);
            int d_bins = dnormal * 1529;
            // float d = depth;
            // if (depth < DMIN) d = DMIN;
            // if (depth > DMAX) d = DMAX;

            // Gonna make it between 0-1 for now
            printf("Running with depth: %d normalized to %f\n", depth, dnormal);
            
            // Convert HSV to RGB
            // hsv in = {dnormal * 360, 1, 1};
            
            // double      hh, p, q, t, ff;
            // long        i;
            // rgb         out;

            // if(in.s <= 0.0) {       // < is bogus, just shuts up warnings
            //     out.r = in.v;
            //     out.g = in.v;
            //     out.b = in.v;
            //     return 0;
            // }
            // hh = in.h;
            // if(hh >= 360.0) hh = 0.0;
            // hh /= 60.0;
            // i = (long)hh;
            // ff = hh - i;
            // p = in.v * (1.0 - in.s);
            // q = in.v * (1.0 - (in.s * ff));
            // t = in.v * (1.0 - (in.s * (1.0 - ff)));

            // switch(i) {
            // case 0:
            //     out.r = in.v;
            //     out.g = t;
            //     out.b = p;
            //     break;
            // case 1:
            //     out.r = q;
            //     out.g = in.v;
            //     out.b = p;
            //     break;
            // case 2:
            //     out.r = p;
            //     out.g = in.v;
            //     out.b = t;
            //     break;

            // case 3:
            //     out.r = p;
            //     out.g = q;
            //     out.b = in.v;
            //     break;
            // case 4:
            //     out.r = t;
            //     out.g = p;
            //     out.b = in.v;
            //     break;
            // case 5:
            // default:
            //     out.r = in.v;
            //     out.g = p;
            //     out.b = q;
            //     break;
            // }
            // uint8_t derp = -1;
            // printf("derp: %d\n", derp);
            // uint8_t derp2 = 256;
            // printf("derp2: %d\n", derp2);
            // uint8_t derp3 = 256 +  256 + 12;
            // printf("derp3: %d\n", derp3);
            // return (int)(out.r * 255) << 16 | (int)(out.g * 255) << 8 | (int)(out.b * 255);
            
            // My sus implementation
            // float H = dnormal * 360;
            
            // float S = 1;
            // float V = 1;
            // float M = 255 * V;
            // float m = M * (1 - S);
            // float z = (M - m) * abs(H / 60 - (int)(H / 60) + ((int)(H / 60) % 2) - 1);
            
            // int to_ret;
            // if (H < 60) {
            //     to_ret = (int)M << 16 | (int)(z + m) << 8 | (int)m;
            // } else if (H < 120) {
            //     to_ret = (int)(z + m) << 16 | (int)M << 8 | (int)m;
            // } else if (H < 180) {
            //     to_ret = (int)m << 16 | (int)M << 8 | (int)(z + m);
            // } else if (H < 240) {
            //     to_ret = (int)m << 16 | (int)(z+m) << 8 | (int)M;
            // } else if (H < 300) {
            //     to_ret = (int)(z + m) << 16 | (int)m << 8 | (int)M;
            // } else {
            //     to_ret = (int)M << 16 | (int)m << 8 | (int)(z + m);
            // }
            // printf("depth %d is mapped to hue: %f\n", depth, hsv.h);
            
            // return 0;
            // return depth;
            // return to_ret;

            int32_t pr = 0;
            if ((0 <= d_bins && d_bins <= 255) || (1275 < d_bins && d_bins <= 1529)) {
                pr = 255;
            } else if (255 < d_bins && d_bins <= 510) {
                pr = 255 - d_bins;
            } else if ( 510 < d_bins && d_bins<= 1020) {
                pr = 0;
            }
            else if ( 1020 < d_bins && d_bins <= 1275) {
                pr = d_bins - 1020;
            }
            else {
                printf("pr not set with d_bins: %d\n", d_bins);
            }

            int32_t pg = 50;
            if (0 <= d_bins && d_bins <= 255) {
                pg = d_bins;
            }
            else if (255 < d_bins && d_bins <= 510) {
                pg = 255;
            }
            else if (510 < d_bins && d_bins <= 765) {
                pg = 765 - d_bins;
            }
            else if (765 < d_bins && d_bins <= 1529) {
                pg = 0;
            }
            else {
                printf("pg not set with d_bins: %d\n", d_bins);
            }

            int32_t pb = 50;
            if (0 <= d_bins && d_bins <= 765) {
                pb = d_bins;
            }
            else if (765 < d_bins && d_bins <= 1020) {
                pb = 765 - d_bins;
            }
            else if (1020 < d_bins && d_bins <= 1275) {
                pb = 255;
            }
            else if (1275 < d_bins && d_bins <= 1529) {
                pb = 1529 - d_bins;
            }
            else {
                printf("pb not set with d_bins: %d\n", d_bins);
            }
            return (pr & 0xFF) << 16 | (pg & 0xFF) << 8 | (pb & 0xFF);
            // pr = 1;
            // pg = 2;
            // pb = 3;

            // RGBA
            // int to_ret = 0;
            // to_ret |= pr;
            // to_ret <<= 8;
            // to_ret |= pg;
            // to_ret <<= 8;
            // to_ret |= pb;
            // to_ret <<= 8;
            // to_ret |= 255;
            // return to_ret;
        }

};


int* colorize(const uint16_t* input, size_t len, float dmin, float dmax) {
    // Depth 16 
    thrust::host_vector<short> h_input(len);
    for (int i = 0; i < len; i++) {
        h_input[i] = input[i];
    }
    std::cout << "input: " << h_input[0] << " " << h_input[1] << " " << h_input[2] << " " << h_input[3] << std::endl;
    thrust::device_vector<short> d_input(len);
    d_input = h_input;
    std::cout << "dinput: " << d_input[0] << " " << d_input[1] << " " << d_input[2] << " " << d_input[3] << std::endl;
    // Output RGBA
    thrust::device_vector<int> d_output(len);
    thrust::transform(d_input.begin(), d_input.end(), d_output.begin(), colorize_functor(dmin, dmax));
    std::cout << "doutput: " << d_output[0] << " " << d_output[1] << " " << d_output[2] << " " << d_output[3] << std::endl;
    thrust::host_vector<int> h_output(len);
    std::cout << "houtput: " << h_output[0] << " " << h_output[1] << " " << h_output[2] << " " << h_output[3] << std::endl;
    thrust::copy(d_output.begin(), d_output.end(), h_output.begin());

    int* output = new int[len];
    for (int i = 0; i < len; i++) {
        output[i] = h_output[i];
    }
    return output;
}





uint16_t to_depth(int rgb_int) {
    // Not sure if the orders are right
    uint8_t r = (rgb_int >> 16) & 0xFF;
    uint8_t g = (rgb_int >> 8) & 0xFF;
    uint8_t b = rgb_int & 0xFF;

    rgb in = {r / 255.0, g / 255.0, b / 255.0};
    hsv out = rgb2hsv(in);
    float H_norm = out.h / 360;

    // uint8_t M = std::max(r, std::max(g, b));
    // uint8_t m = std::min(r, std::min(g, b));

    // // float V = M / 255.0;
    // // float S = 0;
    // // if (M > 0) {
    // //     S = (M - m) / (float)M;
    // // }
    // float H = 0;
    // if (r >= b) {
    //     H = acos((r - 0.5 * g - 0.5 * b) / sqrt(r * r + g * g + b * b - r * g - r * b - g * b));
    // }
    // else {
    //     H = 360 - acos((r - 0.5 * g - 0.5 * b) / sqrt(r * r + g * g + b * b - r * g - r * b - g * b));
    // }
    // float H_norm = H / 360;
    printf("r:%d, g:%d, b:%d -> H_norm: %f\n",r,g,b, H_norm);
    return 0; // UNUSED
    // return DMIN + (DMAX - DMIN) * H_norm;
}

uint16_t to_depth_paper(int rgb_int, float dmin, float dmax) {
    int r = (rgb_int >> 16) & 0xFF;
    int g = (rgb_int >> 8) & 0xFF;
    int b = rgb_int & 0xFF;
    int drnormal;
    if (r >= g && r >= b && g >= b) {
        drnormal = g - b;
    }
    else if (r >= g && r >= b && g < b) {
        drnormal = g - b + 1529;
    }
    else if (g >= r && g >= b) {
        drnormal = b - r + 510;
    }
    else if (b >= g && b >= r){
        drnormal = r - g + 1020;
    }
    else {
        printf("OIWEJFOIJWEOFIEJWOJ Bad color: %d, %d, %d", r, g, b);
    }
    float disp_min = 1/dmax;
    float disp_max = 1/dmin;
    float recovered_depth = 1529.0 / (1529.0 * disp_min + (disp_max - disp_min) * drnormal);

    return recovered_depth;
}

uint8_t* exported_colorize(uint8_t* depth_buf, uint32_t depth_buf_len, float dmin, float dmax) {
    return (uint8_t*)colorize((uint16_t*)depth_buf, depth_buf_len / 2, dmin, dmax);
}

uint8_t* test(uint8_t* arr, uint32_t len) {
    for (int i = 0; i < len; i++) {
        std::cout << (int)arr[i] << std::endl;
        arr[i] += 1;
    }
    return arr;
}

int main() {
    uint16_t input[8] = {300, 301, 302, 310, 500, 1000, 6400, 12800 };
    float dmin = 250;
    float dmax = 5000;
    int* output = colorize(input, 8, dmin, dmax);
    for (int i = 0; i < 8; i++) {
        std::cout << "r:" << (output[i] >> 16 & 0xFF) << " g:" << (output[i] >> 8 & 0xFF) << " b:" << (output[i] & 0xFF) << std::endl;
    }
    // Convert back to depth
    for (int i = 0; i < 8; i++) {
        std::cout << to_depth_paper(output[i], dmin, dmax) << std::endl;
    }
    return 0;
}